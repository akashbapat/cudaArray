// CudaArray: header-only library for interfacing with CUDA array-type objects
// Author: True Price <jtprice at cs.unc.edu>
//
// BSD License
// Copyright (C) 2017  The University of North Carolina at Chapel Hill
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// * Redistributions of source code must retain the above copyright notice, this
//   list of conditions and the following disclaimer.
// * Redistributions in binary form must reproduce the above copyright notice,
//   this list of conditions and the following disclaimer in the documentation
//   and/or other materials provided with the distribution.
// * Neither the name of the original author nor the names of contributors may
//   be used to endorse or promote products derived from this software without
//   specific prior written permission.
//
// NO EXPRESS OR IMPLIED LICENSES TO ANY PARTY'S PATENT RIGHTS ARE GRANTED BY
// THIS LICENSE. THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND
// CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT
// NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
// PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS;
// OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY,
// WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR
// OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF
// ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

#ifndef CUDA_RANDOM_TEST_H_
#define CUDA_RANDOM_TEST_H_

#include "cudaRandomStateArray2D.h"

#include <vector>

#include "gtest/gtest.h"

#include "cudaArray2D.h"
#include "cudaSurface2D.h"
#include "util.h"

//------------------------------------------------------------------------------

template <typename CudaArrayType>
class CudaRandomArray2DTest
    : public PrimitiveConverter<typename CudaArrayType::Scalar> {
 public:
  typedef typename CudaArrayType::Scalar Scalar;
  using PrimitiveConverter<Scalar>::AsScalar;

  //----------------------------------------------------------------------------

  CudaRandomArray2DTest(size_t width = 100, size_t height = 100,
                        size_t seed = 0)
      : array_(width, height),
        random_state_((width + array_.BlockDim().x - 1) / array_.BlockDim().x,
                      (height + array_.BlockDim().y - 1) / array_.BlockDim().y,
                      seed) {}

  //----------------------------------------------------------------------------
  
  template <typename RandomFunction>
  void CheckFillRandom(RandomFunction func, bool check_result = false) {
    array_.Fill(AsScalar(1));
    CUDA_CHECK_ERROR
    array_.FillRandom(random_state_, func);
    CUDA_CHECK_ERROR

    // Check that numbers from the uniform distribution are in [0,1) -- assume
    // that the given random seed never results in exactly 0 being returned,
    // though.
    if (check_result) {
      std::vector<Scalar> result(array_.Size());
      array_.CopyTo(result.data());
      CUDA_CHECK_ERROR

      for (size_t y = 0; y < array_.Height(); ++y) {
        for (size_t x = 0; x < array_.Width(); ++x) {
          const size_t i = y * array_.Width() + x;
          EXPECT_TRUE(All(result[i] > AsScalar(0))) << "Coordinate: " << x
                                                    << " " << y << std::endl
                                                    << "Value: " << result[i];
          EXPECT_TRUE(All(result[i] < AsScalar(1))) << "Coordinate: " << x
                                                    << " " << y << std::endl
                                                    << "Value: " << result[i];
        }
      }
    }
  }

  //----------------------------------------------------------------------------

 private:
  CudaArrayType array_;
  cua::CudaRandomStateArray2D random_state_;
};

//
// Due to the test implementation, our lambda functions need to be defined
// outside of the TEST().
//

template <typename CudaArrayType>
void TestUniform() {
  auto func = [] __device__(hiprandState_t * state) {
    return hiprand_uniform(state);
  };
  CudaRandomArray2DTest<CudaArrayType>().CheckFillRandom(func, true);
}

void TestUniformDouble() {
  auto func = [] __device__(hiprandState_t * state) {
    return hiprand_uniform_double(state);
  };
  CudaRandomArray2DTest<cua::CudaArray2D<double>>().CheckFillRandom(func, true);
}

template <typename CudaArrayType>
void TestLogNormal(float mean, float stddev) {
  auto func = [=] __device__(hiprandState_t * state) {
    return hiprand_log_normal(state, mean, stddev);
  };
  CudaRandomArray2DTest<CudaArrayType>().CheckFillRandom(func);
}

template <typename CudaArrayType>
void TestUnsignedInt() {
  auto func = [] __device__(hiprandState_t * state) { return hiprand(state); };
  CudaRandomArray2DTest<CudaArrayType>().CheckFillRandom(func);
}

//------------------------------------------------------------------------------
//
// Test instances
//
//------------------------------------------------------------------------------

TEST(RandomTest, UnsignedInt) {
  TestUnsignedInt<cua::CudaArray2D<unsigned int>>();
  TestUnsignedInt<cua::CudaSurface2D<unsigned int>>();
}

TEST(RandomTest, UniformFloat) {
  TestUniform<cua::CudaArray2D<float>>();
  TestUniform<cua::CudaSurface2D<float>>();
}

TEST(RandomTest, UniformDouble) {
  TestUniformDouble();
}

TEST(RandomTest, LogNormal) {
  TestLogNormal<cua::CudaArray2D<float>>(0.f, 1.f);
  TestLogNormal<cua::CudaSurface2D<float>>(0.f, 1.f);
  TestLogNormal<cua::CudaArray2D<float>>(2.f, 4.f);
  TestLogNormal<cua::CudaSurface2D<float>>(2.f, 4.f);
}

#endif  // CUDA_RANDOM_TEST_H_
